
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <chrono>

using namespace std;

constexpr long WIDTH = 1024;
constexpr long TILE_WIDTH = 16;

void MatmulOnCPU(double* mat, double* vec, double* P) { 
    for (int i = 0; i < WIDTH; ++i) {
        for (int j = 0; j < WIDTH; ++j){
            double sum = 0;
            sum += mat[i * WIDTH + j] * vec[j];
            P[j] = sum;
        }
    }
}

__global__ void MatrixMulKernel(double *matd, double *vecd, double *Pd);

void MatmulOnGPU(double* mat, double* vec, double* P) {
    constexpr long size = WIDTH * WIDTH;
    double *matd, *vecd, *Pd;
    dim3 dimBlock(TILE_WIDTH);
    dim3 dimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);

    hipMalloc(&matd, size * sizeof(double));
    hipMemcpy(matd, mat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&vecd, WIDTH * sizeof(double));
    hipMemcpy(vecd, vec, WIDTH * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&Pd, WIDTH * sizeof(double));

    MatrixMulKernel <<<dimGrid, dimBlock>>> (matd, vecd, Pd);
    hipMemcpy(P, Pd, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(matd);
    hipFree(vecd);
    hipFree(Pd);
}

__global__ void MatrixMulKernel(double *matd, double *vecd, double *Pd) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    double p = 0;

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) {
        // get the start position of sub-matrix
        auto submatd = matd + by * TILE_WIDTH * WIDTH + m * TILE_WIDTH;
        auto subvecd = vecd + m * TILE_WIDTH * WIDTH + bx * TILE_WIDTH;

        // __shared__ double submatds[TILE_WIDTH][TILE_WIDTH];
        __shared__ double subvecds[TILE_WIDTH];

        // each thread load an element from global memory to shared memory
        // submatds[ty][tx] = submatd[ty * WIDTH + tx];
        subvecds[tx] = subvecd[tx];
        
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            p += submatd[ty * WIDTH + tx] * subvecds[k];
        }

        __syncthreads();
    }
    Pd[tx] = p;

}

int main() {
    assert(WIDTH % TILE_WIDTH == 0);

    constexpr long size = WIDTH * WIDTH;
    double *mat = new double[size];
    double *vec = new double[WIDTH];
    for (int i = 0; i < size; i++) {
        mat[i] = i;
        vec[i] = i;
    }
    double *PCPU = new double[WIDTH];
    double *PGPU = new double[WIDTH];

    chrono::system_clock::time_point begin, end;
    begin = chrono::system_clock::now();
    MatmulOnCPU(mat, vec, PCPU);
    end = chrono::system_clock::now();
    auto cpu_duration = chrono::duration_cast<chrono::microseconds>(end - begin).count();
    
    begin = chrono::system_clock::now();
    MatmulOnGPU(mat, vec, PGPU);
    end = chrono::system_clock::now();
    auto gpu_duration = chrono::duration_cast<chrono::microseconds>(end - begin).count();
    

#ifdef DEBUG
    for (int i = 0; i < WIDTH; i++) {
        printf("%.2lf\t", PCPU[i]);
    }
    for (int i = 0; i < WIDTH; i++) {
        printf("%.2lf\t", PGPU[i]);
    }
#endif
    bool correct = true;
    for (long i = 0; i < size; i++) {
        if (abs(PCPU[i] - PGPU[i]) > 1e-4) {
            correct = false;
            printf("at i = %ld, %lf -- %lf -- %lf\n", i, PCPU[i], PGPU[i], PCPU[i] - PGPU[i]);
            // break;
        }
    }

    printf("=====================Summary=======================\n");
    if (correct) {
        printf("\033[1;32mThe result is correct!\033[0m\n");
    }
    else {
        printf("\033[1;31mThe result is wrong!\033[0m\n");
    }
    printf("cpu:\t %lld us\n", cpu_duration);
    printf("gpu:\t %lld us\n", gpu_duration);
    printf("speedup:\t %lf\n", cpu_duration / (double)gpu_duration);
    printf("===================================================\n");
}